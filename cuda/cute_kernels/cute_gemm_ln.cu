#include "hip/hip_runtime.h"
// #include <hip/hip_cooperative_groups.h>
#include <hip/hip_runtime.h>
#include <cuda/pipeline>
#include <hip/hip_fp16.h>

#include "cute/tensor.hpp"

#include "cuda_utils.h"
#include "gemm_utils.h"

double test_pipeline(std::function<void()> func, const std::string &name,
                     int repeat = -1);

namespace parallel_kernels {

using namespace cute;
using namespace cutlass;
using fp16 = cute::half_t;

extern __shared__ uint8_t shmem_ptr[];

constexpr int kThreadGroupSize = 32 * 4;

static __global__ void __launch_bounds__(kThreadGroupSize)
    kernel_cute_parallel_gemm_ln(fp16 *gemmA_ptr, fp16 *gemmB_ptr,
                                 fp16 *gemmC_ptr, int gemmM, int gemmN,
                                 int gemmK, fp16 *lnA_ptr, fp16 *lnB_ptr,
                                 int lnM) {
  const int thridx = threadIdx.x + threadIdx.y * blockDim.x;
  const int ctaIdx = blockIdx.x + blockIdx.y * gridDim.x;

  using gemmTileM = _128;
  using gemmTileN = _128;
  using gemmTileK = _32;
  using gemmPipe = _4;
  int mtileM = 8;
  int mtileSize = mtileM * gridDim.y;
  int mtileIdx = ctaIdx / mtileSize;
  int tileIdx = ctaIdx % mtileSize;
  int mtileMSize = min(mtileM, gridDim.x - mtileIdx * mtileM);
  int mIdx = mtileIdx * mtileM + tileIdx % mtileMSize;
  int nIdx = tileIdx / mtileMSize;
  auto ctaCoord = make_coord(mIdx, nIdx, _);
  auto gemmTiler = Shape<gemmTileM, gemmTileN, gemmTileK>{};

  auto tensorA =
      make_tensor(make_gmem_ptr(gemmA_ptr),
                  make_layout(make_shape(gemmM, gemmK), LayoutRight{}));
  auto tensorB =
      make_tensor(make_gmem_ptr(gemmB_ptr),
                  make_layout(make_shape(gemmM, gemmK), LayoutRight{}));
  auto tensorC =
      make_tensor(make_gmem_ptr(gemmC_ptr),
                  make_layout(make_shape(gemmM, gemmN), LayoutRight{}));
  using AsyncCopy =
      Copy_Atom<Copy_Traits<SM80_CP_ASYNC_CACHEGLOBAL<uint128_t>>, fp16>;

  Tensor tAGmem = local_tile(tensorA, gemmTiler, ctaCoord, Step<_1, X, _1>{});
  Tensor tBGmem = local_tile(tensorB, gemmTiler, ctaCoord, Step<X, _1, _1>{});
  Tensor tCGmem = local_tile(tensorC, gemmTiler, ctaCoord, Step<_1, _1, X>{});

  auto layoutASmem = composition(
      Swizzle<3, 3, 3>{},
      make_layout(Shape<gemmTileM, gemmTileK, gemmPipe>{},
                  make_stride(gemmTileK{}, _1{}, gemmTileM{} * gemmTileK{})));
  auto layoutBSmem = composition(
      Swizzle<3, 3, 3>{},
      make_layout(Shape<gemmTileN, gemmTileK, gemmPipe>{},
                  make_stride(gemmTileK{}, _1{}, gemmTileN{} * gemmTileK{})));
  auto layoutCSmem =
      composition(Swizzle<3, 4, 3>{},
                  make_layout(Shape<gemmTileM, gemmTileN>{}, LayoutRight{}));
  fp16 *sA = (fp16 *)shmem_ptr;
  fp16 *sB = sA + cosize_v<decltype(layoutASmem)>;
  fp16 *sC = (fp16 *)shmem_ptr; // sA is out of scope when sC is alive
  Tensor tASmem = make_tensor(make_smem_ptr(sA), layoutASmem);
  Tensor tBSmem = make_tensor(make_smem_ptr(sB), layoutBSmem);
  Tensor tCSmem = make_tensor(make_smem_ptr(sC), layoutCSmem);

  auto copyA =
      make_tiled_copy(AsyncCopy{}, make_layout(Shape<_32, _4>{}, LayoutRight{}),
                      make_layout(Shape<_1, _8>{}, LayoutRight{}));
  auto copyB = copyA;

  ThrCopy thrCopyA = copyA.get_slice(thridx);
  Tensor copyASrc = thrCopyA.partition_S(tAGmem);
  Tensor copyADst = thrCopyA.partition_D(tASmem);
  ThrCopy thrCopyB = copyB.get_slice(thridx);
  Tensor copyBSrc = thrCopyB.partition_S(tBGmem);
  Tensor copyBDst = thrCopyB.partition_D(tBSmem);

  auto blockMMA = make_tiled_mma(
      MMA_Atom<MMA_Traits<SM80_16x8x16_F32F16F16F32_TN>>{},
      make_layout(Shape<_2, _2>{}, LayoutRight{}),
      Tile<decltype(get<0>(gemmTiler)), decltype(get<1>(gemmTiler)), _16>{});
  ThrMMA thrMMA = blockMMA.get_slice(thridx);
  Tensor mmaASmem = thrMMA.partition_A(tASmem);
  Tensor mmaBSmem = thrMMA.partition_B(tBSmem);
  Tensor mmaCSmem = thrMMA.partition_C(tCSmem);

  Tensor mmaAReg = thrMMA.make_fragment_A(mmaASmem(_, _, _, 0));
  Tensor mmaBReg = thrMMA.make_fragment_B(mmaBSmem(_, _, _, 0));
  Tensor mmaCReg = thrMMA.make_fragment_C(mmaCSmem);

  clear(mmaCReg);
  using LdMatrix = Copy_Atom<Copy_Traits<SM75_U32x4_LDSM_N>, fp16>;

  auto ldmA = make_tiled_copy_A(LdMatrix{}, blockMMA);
  auto thrLdmA = ldmA.get_slice(thridx);
  auto ldmASrc = thrLdmA.partition_S(tASmem);
  auto ldmADst = thrLdmA.retile_D(mmaAReg);
  auto ldmB = make_tiled_copy_B(LdMatrix{}, blockMMA);
  auto ldmThrB = ldmB.get_slice(thridx);
  auto ldmBSrc = ldmThrB.partition_S(tBSmem);
  auto ldmBDst = ldmThrB.retile_D(mmaBReg);

  constexpr int kMaxPipe = size<3>(mmaASmem);
  int kTileCount = size<3>(copyASrc);
  int kMaxBlock = size<2>(mmaASmem);
  int loadIdx = 0;

  int usePrologue = 1; // 0 or 1
  if (usePrologue) {
    int pipeIdx = 0;
    int mmaIdx = 0;
    copy(copyA, copyASrc(_, _, _, pipeIdx), copyADst(_, _, _, loadIdx));
    copy(copyB, copyBSrc(_, _, _, pipeIdx), copyBDst(_, _, _, loadIdx));
    cp_async_fence();
    cp_async_wait<0>();
    __syncthreads();
    CUTE_UNROLL
    for (int blockIdx = 0; blockIdx < kMaxBlock; blockIdx++) {
      copy(ldmA, ldmASrc(_, _, blockIdx, mmaIdx), ldmADst(_, _, blockIdx));
      copy(ldmB, ldmBSrc(_, _, blockIdx, mmaIdx), ldmBDst(_, _, blockIdx));
      gemm(blockMMA, mmaAReg(_, _, blockIdx), mmaBReg(_, _, blockIdx), mmaCReg);
    }
    loadIdx = 1;
  }

  for (int pipeIdx = usePrologue; pipeIdx < kTileCount + kMaxPipe - 1;
       pipeIdx++) {
    if (pipeIdx < kTileCount) {
      copy(copyA, copyASrc(_, _, _, pipeIdx), copyADst(_, _, _, loadIdx));
      copy(copyB, copyBSrc(_, _, _, pipeIdx), copyBDst(_, _, _, loadIdx));
      cp_async_fence();
    }
    // if (kMaxPipe - 1 <= pipeIdx - usePrologue) {
    // }
    if (kMaxPipe - 1 <= pipeIdx - usePrologue) {
      cp_async_wait<kMaxPipe - 1>();
      __syncthreads();
      int mmaIdx = loadIdx + 1;
      mmaIdx = mmaIdx == kMaxPipe ? 0 : mmaIdx;
      CUTE_UNROLL
      for (int blockIdx = 0; blockIdx < kMaxBlock; blockIdx++) {
        copy(ldmA, ldmASrc(_, _, blockIdx, mmaIdx), ldmADst(_, _, blockIdx));
        copy(ldmB, ldmBSrc(_, _, blockIdx, mmaIdx), ldmBDst(_, _, blockIdx));
        gemm(blockMMA, mmaAReg(_, _, blockIdx), mmaBReg(_, _, blockIdx),
             mmaCReg);
      }
    }
    loadIdx += 1;
    loadIdx = loadIdx == kMaxPipe ? 0 : loadIdx;
  }

  Tensor mmaCRegFp16 = make_fragment_like<fp16>(mmaCReg.layout());
  using StMatrix = Copy_Atom<UniversalCopy<AlignedArray<fp16, 2>>, fp16>;
  using Copy8B = Copy_Atom<UniversalCopy<AlignedArray<fp16, 4>>, fp16>;
  auto stmC = make_tiled_copy_C(StMatrix{}, blockMMA);
  auto copyC =
      make_tiled_copy(Copy8B{}, make_layout(Shape<_4, _32>{}, LayoutRight{}),
                      make_layout(Shape<_1, _4>{}, LayoutRight{}));
  auto thrCopyC = copyC.get_slice(thridx);
  auto copyCSrc = thrCopyC.partition_S(tCSmem);
  auto copyCDst = thrCopyC.partition_D(tCGmem);
  for (int j = 0; j < size<2>(mmaCReg); j++) {
    auto regFp16 = mmaCRegFp16(_, _, j);
    auto regFp32 = mmaCReg(_, _, j);
    auto smem = mmaCSmem(_, _, j);
    CUTE_UNROLL
    for (int i = 0; i < size(regFp32); i += 2) {
      __half2 packedHalf = __floats2half2_rn(regFp32[i], regFp32[i + 1]);
      regFp16[i] = packedHalf.x;
      regFp16[i + 1] = packedHalf.y;
    }
    copy(stmC, regFp16, smem);
  }
  __syncthreads();
  copy(copyC, copyCSrc, copyCDst);

  // int total_workers = gridDim.x * gridDim.y;
  // int start_idx = (ctaIdx * 128 + thridx - 128) * 2;
  // int stride = total_workers * 128 * 2;
  // int step = 0;
  // for (int i = start_idx; i < lnM; i += stride) {
  //   if (step <= gemmK / 32) {
  //     __syncthreads();
  //   }
  //   __half2 tmp = *reinterpret_cast<__half2 *>(&lnA_ptr[i]);
  //   __half2 tmp2;
  //   tmp2.x = tmp.x / (__half(1.0) + hexp(-tmp.x));
  //   tmp2.y = tmp.y / (__half(1.0) + hexp(-tmp.y));
  //   *reinterpret_cast<__half2 *>(&lnB_ptr[i]) = tmp2;
  //   step++;
  // }
  // while (step <= gemmK / 32) {
  //   __syncthreads();
  //   step++;
  // }
}

void entry_cute_parallel_gemmrc_lnr(fp16 *gemmA_ptr, fp16 *gemmB_ptr,
                                    fp16 *gemmC_ptr, int gemmM, int gemmN,
                                    int gemmK, fp16 *lnA_ptr, fp16 *lnB_ptr,
                                    int lnM, int lnN) {
  dim3 gridDim(gemmM / 128, gemmN / 128);
  dim3 blockDim(kThreadGroupSize, 1, 1);

  constexpr int SMEM_SIZE = 64 * 1024;
  cudaSafeCall(hipFuncSetAttribute(reinterpret_cast<const void*>(
      parallel_kernels::kernel_cute_parallel_gemm_ln),
      hipFuncAttributeMaxDynamicSharedMemorySize, SMEM_SIZE));
  std::string name = "cutlass_parallel_gemmrc";
  // int lnSize = lnM * lnN;
  // void *kernelArgs[] = {(void *)gemmA_ptr, (void *)gemmB_ptr, (void
  // *)gemmC_ptr,
  //                       (void *)&gemmM,    (void *)&gemmN,    (void *)&gemmK,
  //                       (void *)lnA_ptr,   (void *)lnB_ptr,   (void
  //                       *)&lnSize};
  double latency = test_pipeline(
      [&]() {
        // hipLaunchCooperativeKernel(
        //     (void *)parallel_kernels::kernel_cute_parallel_gemm_ln, gridDim,
        //     blockDim, kernelArgs, SMEM_SIZE, nullptr);
        parallel_kernels::
            kernel_cute_parallel_gemm_ln<<<gridDim, blockDim, SMEM_SIZE>>>(
                gemmA_ptr, gemmB_ptr, gemmC_ptr, gemmM, gemmN, gemmK, lnA_ptr,
                lnB_ptr, lnM * lnN);
      },
      name);
  double tflops = get_tflops(gemmM, gemmN, gemmK, latency);
  printf("%s: %.2f TFLOPS\n", name.data(), tflops);
}

} // namespace parallel_kernels
