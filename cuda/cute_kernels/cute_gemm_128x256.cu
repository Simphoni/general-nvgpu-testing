#include "hip/hip_runtime.h"
#include "cute/tensor.hpp"

#include "cuda_utils.h"
#include "gemm_utils.h"

namespace parallel_kernels {

using namespace cute;
using namespace cutlass;
using fp16 = cute::half_t;

extern __shared__ uint8_t shmem_ptr[];

static constexpr int algoTileM = 128;
static constexpr int algoTileN = 256;

static __global__ void __launch_bounds__(256)
    kernel_cute_parallel_gemm(fp16 *gemmA_ptr, fp16 *gemmB_ptr,
                              fp16 *gemmC_ptr, // fp16 *lnA, fp16 *lnB,
                              int gemmM, int gemmN, int gemmK) {
  using gemmTileM = C<algoTileM>;
  using gemmTileN = C<algoTileN>;
  using gemmTileK = _32;
  using gemmPipe = _6;
  const int thridx = threadIdx.x + threadIdx.y * blockDim.x;
  int ctaIdx = blockIdx.x + blockIdx.y * gridDim.x;
  // mtile for 'Microtile'.
  // It's used for mapping between CTA and the Gemm sub-matrix,
  // targeting better L2 cache utilization.
  constexpr int mtileM = 16;
  int mtileSize = mtileM * gridDim.y;
  int mtileIdx = ctaIdx / mtileSize;
  int tileIdx = ctaIdx % mtileSize;
  int mtileMSize = min(mtileM, gridDim.x - mtileIdx * mtileM);
  int mIdx = mtileIdx * mtileM + tileIdx % mtileMSize;
  int nIdx = tileIdx / mtileMSize;
  auto ctaCoord = make_coord(mIdx, nIdx, _);
  // auto ctaCoord = make_coord(blockIdx.x, blockIdx.y, _);
  auto gemmTiler = Shape<gemmTileM, gemmTileN, gemmTileK>{};

  auto tensorA =
      make_tensor(make_gmem_ptr(gemmA_ptr),
                  make_layout(make_shape(gemmM, gemmK), LayoutRight{}));
  auto tensorB =
      make_tensor(make_gmem_ptr(gemmB_ptr),
                  make_layout(make_shape(gemmM, gemmK), LayoutRight{}));
  auto tensorC =
      make_tensor(make_gmem_ptr(gemmC_ptr),
                  make_layout(make_shape(gemmM, gemmN), LayoutRight{}));
  // CP_ASYNC only accept 16B, cutlass 3.5.1 assertion too loose
  using AsyncCopy =
      Copy_Atom<Copy_Traits<SM80_CP_ASYNC_CACHEGLOBAL<uint128_t>>, fp16>;

  // when masked, ctaCoord become (blockIdx.x, _), selecting all the column
  Tensor tAGmem = local_tile(tensorA, gemmTiler, ctaCoord, Step<_1, X, _1>{});
  Tensor tBGmem = local_tile(tensorB, gemmTiler, ctaCoord, Step<X, _1, _1>{});
  Tensor tCGmem = local_tile(tensorC, gemmTiler, ctaCoord, Step<_1, _1, X>{});

  auto layoutASmem = composition(
      Swizzle<3, 3, 3>{},
      make_layout(Shape<gemmTileM, gemmTileK, gemmPipe>{},
                  make_stride(gemmTileK{}, _1{}, gemmTileM{} * gemmTileK{})));
  auto layoutBSmem = composition(
      Swizzle<3, 3, 3>{},
      make_layout(Shape<gemmTileN, gemmTileK, gemmPipe>{},
                  make_stride(gemmTileK{}, _1{}, gemmTileN{} * gemmTileK{})));
  auto layoutCSmem =
      composition(Swizzle<3, 4, 3>{},
                  make_layout(Shape<gemmTileM, gemmTileN>{}, LayoutRight{}));
  // [cosize_v<decltype(layoutASmem)>];
  fp16 *sA = (fp16 *)shmem_ptr;
  // [cosize_v<decltype(layoutBSmem)>];
  fp16 *sB = sA + cosize_v<decltype(layoutASmem)>;
  fp16 *sC = (fp16 *)shmem_ptr; // sA is out of scope when sC is alive
  Tensor tASmem = make_tensor(make_smem_ptr(sA), layoutASmem);
  Tensor tBSmem = make_tensor(make_smem_ptr(sB), layoutBSmem);
  Tensor tCSmem = make_tensor(make_smem_ptr(sC), layoutCSmem);

  auto copyA =
      make_tiled_copy(AsyncCopy{}, make_layout(Shape<_64, _4>{}, LayoutRight{}),
                      make_layout(Shape<_1, _8>{}, LayoutRight{}));
  auto copyB = copyA;

  // (COPY,COPY_M,COPY_K,k_tile_count)
  ThrCopy thrCopyA = copyA.get_slice(thridx);
  Tensor copyASrc = thrCopyA.partition_S(tAGmem);
  Tensor copyADst = thrCopyA.partition_D(tASmem);
  ThrCopy thrCopyB = copyB.get_slice(thridx);
  Tensor copyBSrc = thrCopyB.partition_S(tBGmem);
  Tensor copyBDst = thrCopyB.partition_D(tBSmem);

  // thr_layout tiles the work onto all warps
  auto blockMMA =
      make_tiled_mma(MMA_Atom<MMA_Traits<SM80_16x8x16_F32F16F16F32_TN>>{},
                     make_layout(Shape<_2, _4>{}, LayoutRight{}),
                     Tile<gemmTileM, gemmTileN, _16>{});
  ThrMMA thrMMA = blockMMA.get_slice(thridx);
  // partition_A expect (M, K, ...)
  Tensor mmaASmem = thrMMA.partition_A(tASmem);
  // partition_B expect (N, K, ...)
  Tensor mmaBSmem = thrMMA.partition_B(tBSmem);
  // partition_C expect (M, N, ...)
  Tensor mmaCSmem = thrMMA.partition_C(tCSmem);

  Tensor mmaAReg = thrMMA.make_fragment_A(mmaASmem(_, _, _, 0));
  Tensor mmaBReg = thrMMA.make_fragment_B(mmaBSmem(_, _, _, 0));
  Tensor mmaCReg = thrMMA.make_fragment_C(mmaCSmem);

  clear(mmaCReg);
  // this is the ldmatrix 8x8 command (on 4 matrices)
  using LdMatrix = Copy_Atom<Copy_Traits<SM75_U32x4_LDSM_N>, fp16>;

  auto ldmA = make_tiled_copy_A(LdMatrix{}, blockMMA);
  auto thrLdmA = ldmA.get_slice(thridx);
  auto ldmASrc = thrLdmA.partition_S(tASmem);
  auto ldmADst = thrLdmA.retile_D(mmaAReg);
  auto ldmB = make_tiled_copy_B(LdMatrix{}, blockMMA);
  auto ldmThrB = ldmB.get_slice(thridx);
  auto ldmBSrc = ldmThrB.partition_S(tBSmem);
  auto ldmBDst = ldmThrB.retile_D(mmaBReg);

  constexpr int kMaxPipe = size<3>(mmaASmem);
  constexpr int kAsyncCopy = kMaxPipe - 2;
  int kTileCount = size<3>(copyASrc);
  int kMaxBlock = size<2>(mmaASmem);

  for (int pipeIdx = 0; pipeIdx < kMaxPipe - 1; pipeIdx++) {
    copy(copyA, copyASrc(_, _, _, pipeIdx), copyADst(_, _, _, pipeIdx));
    copy(copyB, copyBSrc(_, _, _, pipeIdx), copyBDst(_, _, _, pipeIdx));
    cp_async_fence();
  }

  int loadIdx = kMaxPipe - 1;
  int mmaIdx = 0;
  static_assert(kAsyncCopy > 0);
  cp_async_wait<kAsyncCopy>();
  __syncthreads();
  copy(ldmA, ldmASrc(_, _, 0, mmaIdx), ldmADst(_, _, 0));
  copy(ldmB, ldmBSrc(_, _, 0, mmaIdx), ldmBDst(_, _, 0));

  CUTE_NO_UNROLL
  for (int pipeIdx = kMaxPipe - 1; pipeIdx < kTileCount + kMaxPipe - 1;
       pipeIdx++) {
    CUTE_UNROLL
    for (int blockIdx = 0; blockIdx < kMaxBlock; blockIdx++) {
      if (blockIdx == kMaxBlock - 1) {
        cp_async_wait<kAsyncCopy>();
        __syncthreads();
        // SRAM->Reg works on a new stage
        mmaIdx += 1;
        mmaIdx = mmaIdx == kMaxPipe ? 0 : mmaIdx;
      }
      int copyBlk = blockIdx + 1 == kMaxBlock ? 0 : blockIdx + 1;
      copy(ldmA, ldmASrc(_, _, copyBlk, mmaIdx), ldmADst(_, _, copyBlk));
      copy(ldmB, ldmBSrc(_, _, copyBlk, mmaIdx), ldmBDst(_, _, copyBlk));
      if (blockIdx == 0) {
        if (pipeIdx < kTileCount) {
          copy(copyA, copyASrc(_, _, _, pipeIdx), copyADst(_, _, _, loadIdx));
          copy(copyB, copyBSrc(_, _, _, pipeIdx), copyBDst(_, _, _, loadIdx));
          cp_async_fence();
          loadIdx += 1;
          loadIdx = loadIdx == kMaxPipe ? 0 : loadIdx;
        }
      }
      gemm(blockMMA, mmaAReg(_, _, blockIdx), mmaBReg(_, _, blockIdx), mmaCReg);
    }
  }

  using StMatrix = Copy_Atom<UniversalCopy<AlignedArray<fp16, 2>>, fp16>;
  using Copy16B = Copy_Atom<UniversalCopy<uint128_t>, fp16>;
  auto stmC = make_tiled_copy_C(StMatrix{}, blockMMA);
  auto copyC =
      make_tiled_copy(Copy16B{}, make_layout(Shape<_8, _32>{}, LayoutRight{}),
                      make_layout(Shape<_1, _8>{}, LayoutRight{}));
  auto thrCopyC = copyC.get_slice(thridx);
  auto copyCSrc = thrCopyC.partition_S(tCSmem);
  auto copyCDst = thrCopyC.partition_D(tCGmem);
  if (threadIdx.x == 0 && blockIdx.x == 0 && blockIdx.y == 0) {
  }
  for (int j = 0; j < size<2>(mmaCReg); j++) {
    auto regFp32 = mmaCReg(_, _, j);
    auto regFp16 = make_fragment_like<fp16>(regFp32.layout());
    auto smem = mmaCSmem(_, _, j);
    CUTE_UNROLL
    for (int i = 0; i < size(regFp32); i += 2) {
      __half2 packedHalf = __floats2half2_rn(regFp32[i], regFp32[i + 1]);
      regFp16[i] = packedHalf.x;
      regFp16[i + 1] = packedHalf.y;
    }
    copy(stmC, regFp16, smem);
  }
  __syncthreads();
  copy(copyC, copyCSrc, copyCDst);
}

void entry_custom_gemmrc_128x256(fp16 *gemmA_ptr, fp16 *gemmB_ptr,
                                 fp16 *gemmC_ptr, int gemmM, int gemmN,
                                 int gemmK) {
  dim3 gridDim(gemmM / algoTileM, gemmN / algoTileN);
  dim3 blockDim(256, 1, 1);

  constexpr int smemSize = 144 * 1024;
  cudaSafeCall(hipFuncSetAttribute(reinterpret_cast<const void*>(parallel_kernels::kernel_cute_parallel_gemm),
                                    hipFuncAttributeMaxDynamicSharedMemorySize,
                                    smemSize));
  parallel_kernels::kernel_cute_parallel_gemm<<<gridDim, blockDim, smemSize>>>(
      gemmA_ptr, gemmB_ptr, gemmC_ptr, gemmM, gemmN, gemmK);
}

} // namespace parallel_kernels
