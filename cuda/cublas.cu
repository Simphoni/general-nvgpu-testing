#include <cstdio>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>

#define cudaSafeCall(err) __cudaSafeCall(err, __FILE__, __LINE__)
inline void __cudaSafeCall(hipError_t err, const char *file, const int line) {
    if (hipSuccess != err) {
        fprintf(stderr, "cudaSafeCall() failed at %s:%i : %s\n", file, line, hipGetErrorString(err));
        exit(-1);
    }
}

#define cublasSafeCall(err) __cublasSafeCall(err, __FILE__, __LINE__)
inline void __cublasSafeCall(hipblasStatus_t err, const char *file, const int line) {
    if (HIPBLAS_STATUS_SUCCESS != err) {
        fprintf(stderr, "cublasSafeCall() failed at %s:%i : %d\n", file, line, err);
        exit(-1);
    }
}

#define curandSafeCall(err) __curandSafeCall(err, __FILE__, __LINE__)
inline void __curandSafeCall(hiprandStatus_t err, const char *file, const int line) {
    if (HIPRAND_STATUS_SUCCESS != err) {
        fprintf(stderr, "curandSafeCall() failed at %s:%i : %d\n", file, line, err);
        exit(-1);
    }
}

int main() {
    int n = 4096;
    int m = 4096;
    int k = 4096;
    
}